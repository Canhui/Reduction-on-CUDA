#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

typedef unsigned long long uint64_t;

#define RT 1000 // Kernel repeats 1000 times

void initData(uint64_t *data, uint64_t data_len);
void printData(uint64_t *data, uint64_t data_len, float using_time);
__global__ void reduce_v0(uint64_t *data_gpu);



int main(){
	
	/*--Init Data on Host Momory-------*/
	uint64_t data_len = 1024;
	uint64_t *data = (uint64_t *) malloc(sizeof(uint64_t) * data_len);
	initData(data, data_len);


	/*--Init CUDA Environment----*/
	int threads_num = 1024;
	int blocks_num = 1;
	dim3 threads, blocks;
	threads.x = threads_num;
	blocks.x = blocks_num;
	hipSetDevice(0);
	

	/*--Load Data from Host to Device---*/
	uint64_t * data_gpu;
	hipMalloc((uint64_t**)&data_gpu, sizeof(uint64_t) * data_len);
	hipMemcpy(data_gpu, data, sizeof(uint64_t) * data_len, hipMemcpyHostToDevice);

	/*--Run CUDA Kernel--*/
	float using_time = 0.0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// repeating 1000 times
	for(int i = 0; i < RT; i++){
		reduce_v0<<<blocks, threads>>>(data_gpu);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&using_time, start, stop);

	/*--Store Data from Device to Host--*/
	hipMemcpy(data, data_gpu, sizeof(uint64_t) * 1, hipMemcpyDeviceToHost);
	printData(data, 1, using_time);
	
	
	return 0;
}



void initData(uint64_t *data, uint64_t data_len){
	for(uint64_t i = 0; i < data_len; i++){
		data[i] = i;
	}
}


void printData(uint64_t *data, uint64_t data_len, float using_time){
	printf("\n-----Reduction Result (version 1)----\n");
	printf("\n0+1+2+3+...+1023 = ");
	uint64_t count = 0;
	for(uint64_t i = 0; i < data_len; i++){
		count ++;
		if(count % 11 == 0) printf("\n");
		printf("%llu",data[i]/RT);
	}
	printf("\n\nusing time (repeating %d times): %f(ms)\n", RT, using_time);
	printf("\n\n-----The End--------\n\n");
}


__global__ void reduce_v0(uint64_t *data_gpu){
	int tid = threadIdx.x;

	// load data into shared memory
	__shared__ uint64_t data[1024];
	data[tid] = data_gpu[tid];
	__syncthreads();

	// reduction
	for (int i=1; i < 1024; i *= 2){
        if ((tid % (2 * i)) == 0){
            data[tid] += data[tid + i];
        }
        __syncthreads();
    }

	// write root node (data[0]) back
	if(tid == 0){
		data_gpu[tid] = data[tid];
	}
}